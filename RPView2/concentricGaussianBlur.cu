#include <hip/hip_runtime.h>

#define PI 3.14159265358979323846f

__device__ float gaussian(float x, float sigma) {
    return expf(-(x * x) / (2 * sigma * sigma)) / (sqrtf(2 * PI) * sigma);
}

__global__ void concentricGaussianBlur(unsigned char* input, unsigned char* output, int width, int height, int centerX,
                                       int centerY, float maxRadius, int minKernelSize, int maxKernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    int idx = (y * width + x) * 3;  // �J���[�摜�Ȃ̂ŁA�s�N�Z�����Ƃ�3�`�����l��

    // �������͈̓`�F�b�N
    if (idx + 2 >= width * height * 3) {
        return;  // �������͈͊O�ւ̃A�N�Z�X��h�~
    }

    // R, G, B ��3�̃`�����l��������
    for (int c = 0; c < 3; c++) {
#if 0
        output[idx + c] = input[idx + c];
#else
        float dx = x - centerX;
        float dy = y - centerY;
        float distance = sqrtf(dx * dx + dy * dy);

        // �����Ɋ�Â��ăJ�[�l���T�C�Y������
        int kernelSize;
        if (distance > maxRadius) {
            kernelSize = 1;
        } else {
            // �����Ɋ�Â��ăJ�[�l���T�C�Y����`�ɃX�P�[�����O
            float normalizedDistance = distance / maxRadius;
            kernelSize = minKernelSize + (int)((maxKernelSize - minKernelSize) * normalizedDistance);
        }

        if (kernelSize % 2 == 0) {
            kernelSize++;
        }

        float sigma = kernelSize / 3.0f;
        float sum = 0.0f;
        float weightSum = 0.0f;
        int halfKernel = kernelSize / 2;

        for (int ky = -halfKernel; ky <= halfKernel; ky++) {
            for (int kx = -halfKernel; kx <= halfKernel; kx++) {
                int nx = min(max(x + kx, 0), width - 1);
                int ny = min(max(y + ky, 0), height - 1);
                int neighborIdx = (ny * width + nx) * 3 + c;  // �אڃs�N�Z���̃`�����l�����Q��

                float weight = gaussian(sqrtf(kx * kx + ky * ky), sigma);

                sum += input[neighborIdx] * weight;
                weightSum += weight;
            }
        }

        output[idx + c] = static_cast<unsigned char>(sum / weightSum);  // �e�`�����l��������
#endif
    }
}

// CUDA�J�[�l���̌Ăяo���֐�
extern "C" void runConcentricGaussianBlur(unsigned char* input, unsigned char* output, int width, int height,
                                          int centerX, int centerY, float maxRadius, int minKernelSize,
                                          int maxKernelSize) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    concentricGaussianBlur<<<blocksPerGrid, threadsPerBlock>>>(input, output, width, height, centerX, centerY,
                                                               maxRadius, minKernelSize, maxKernelSize);

    hipDeviceSynchronize();
}
